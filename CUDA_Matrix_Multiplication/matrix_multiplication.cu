#include <stdio.h>
#include <hip/hip_runtime.h>

#define TILE_WIDTH 16

__global__ void matrixMulKernel (float* d_N, float* d_M, float* d_P, int width) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if ((row < width) && (col < width)) {
        float Pvalue = 0;
        for (int k = 0; k < width; ++k) {
            Pvalue += d_N[row*width+k]*d_M[k*width+col];
            d_P[row*width+col] = Pvalue;
        }
    }
}

int main() {
    int width = 32;
    size_t size = width * width * sizeof(float);

    hipEvent_t start, end; 
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start,0);

    float *h_M = (float*)malloc(size);
    float *h_N = (float*)malloc(size);
    float *h_P = (float*)malloc(size);

    float *d_M, *d_N, *d_P;
    hipMalloc((void**)&d_M, size);
    hipMalloc((void**)&d_N, size);
    hipMalloc((void**)&d_P, size);

    hipMemcpy(d_M, h_M, size, hipMemcpyHostToDevice);
    hipMemcpy(d_N, h_N, size, hipMemcpyHostToDevice);

    dim3 dimGrid(ceil(width/(TILE_WIDTH*1.0)), ceil(width/(TILE_WIDTH*1.0)), 1);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);

    matrixMulKernel<<<dimGrid, dimBlock>>>(d_M, d_N, d_P, width);
    
    hipEventRecord(end,0);
    hipEventSynchronize(end);

    hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(error));
		return 1;
	}
    float time_ms = 0;
    hipEventElapsedTime(&time_ms, start, end);

    hipMemcpy(h_P, d_P, size, hipMemcpyDeviceToHost);
    printf("Execution time for kernel: %.2f ms\n", time_ms);
    printf("results: %d\n", h_P);

    free(h_M);
    free(h_N);
    free(h_P);

    hipFree(d_M);
    hipFree(d_N);
    hipFree(d_P);

    return 0;
}