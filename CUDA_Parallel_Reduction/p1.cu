
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ void warpReduce(volatile int* sdata, int tid) {
	sdata[tid] += sdata[tid+32];
	sdata[tid] += sdata[tid+16];
	sdata[tid] += sdata[tid+8];
	sdata[tid] += sdata[tid+4];
	sdata[tid] += sdata[tid+2];
	sdata[tid] += sdata[tid+1];
}

__global__
void reduce0(int *g_idata, int *g_odata) {
	extern __shared__ int sdata[];
	// each thread loads one element from global to shared mem
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
	size_t vector_size = 1 << 20;
	if (i + blockDim.x < vector_size) {
		sdata[tid] = g_idata[i] + g_idata[i + blockDim.x];
	} else if (i < vector_size) {
		sdata[tid] = g_idata[i];
	} else {
		sdata[tid] = 0;
	}
	__syncthreads();
	// do reduction in shared mem
	for(unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
		if (tid < s) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}
	// write result for this block to global mem
	if (tid < 32) warpReduce(sdata, tid);
	if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

int main() {
	int *h_in, h_out, *d_in, *d_out;
	size_t vector_size;
	float time_ms = 0;
	int n_threads = 1024;

	hipEvent_t t1, t2;

	vector_size = 1 << 20;

	h_in = (int*)malloc(sizeof(int) * vector_size);
	hipMalloc((void**)&d_in, sizeof(int) * vector_size);
	hipMalloc((void**)&d_out, sizeof(int) * (vector_size/n_threads + 1));

	for (int i = 0; i < vector_size; i++) {
		h_in[i] = 1;
	}

	hipError_t err = hipMemcpy(d_in, h_in, sizeof(int) * vector_size, hipMemcpyHostToDevice);

	hipEventCreate(&t1);
	hipEventCreate(&t2);

	hipEventRecord(t1, 0);

	reduce0<<<vector_size / n_threads, n_threads, n_threads*sizeof(int)>>>(d_in, d_out);
	reduce0<<<1, n_threads, n_threads*sizeof(int)>>>(d_out, d_out+(vector_size/n_threads));


	hipEventRecord(t2, 0);
	hipEventSynchronize(t2);

	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		// somthing's gone wrong
		// print out the CUDA error as a string
		fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(error));

		// we can't recover from the error -- exit the program
		return 1;
	}

	hipEventElapsedTime(&time_ms, t1, t2);

	hipMemcpy(&h_out, d_out+(vector_size/n_threads), sizeof(int), hipMemcpyDeviceToHost);
	printf("Execution time for reduce0: %.2f ms\n", time_ms);
	printf("results: %d\n", h_out);

	hipFree(d_out);
	hipFree(d_in);
	free(h_in);
	return 0;
}
